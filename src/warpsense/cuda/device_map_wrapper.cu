#include "warpsense/cuda/device_map_wrapper.h"

namespace cuda
{

DeviceMapMemWrapper::DeviceMapMemWrapper(size_t n_voxels)
: n_voxels_(n_voxels)
{
  // allocate inner map
  CHECK(hipMalloc(&inner_.size_, sizeof(rmagine::Pointi)));
  CHECK(hipMalloc(&inner_.offset_, sizeof(rmagine::Pointi)));

  CHECK(hipMalloc(&inner_.data_, n_voxels * sizeof(TSDFEntry)));
  CHECK(hipMalloc(&inner_.pos_, sizeof(rmagine::Pointi)));

  // allocate device pointer
  CHECK(hipMalloc((void**) &dev_, sizeof(cuda::DeviceMap)));
}

DeviceMapMemWrapper::DeviceMapMemWrapper(const DeviceMap& existing_map)
: DeviceMapMemWrapper(existing_map.size_->prod())
{
  to_device(existing_map);
}

DeviceMapMemWrapper::~DeviceMapMemWrapper()
{
  CHECK(hipFree(inner_.size_));
  CHECK(hipFree(inner_.offset_));
  CHECK(hipFree(inner_.data_));
  CHECK(hipFree(inner_.pos_));
  CHECK(hipFree(dev_));
}

void DeviceMapMemWrapper::to_device(const cuda::DeviceMap &existing_map)
{
  // copy inner map
  CHECK(hipMemcpy(inner_.size_, existing_map.size_, sizeof(rmagine::Pointi), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(inner_.offset_, existing_map.offset_, sizeof(rmagine::Pointi), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(inner_.data_, existing_map.data_, n_voxels_ * sizeof(TSDFEntry), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(inner_.pos_, existing_map.pos_, sizeof(rmagine::Pointi), hipMemcpyHostToDevice));

  // copy device pointer
  CHECK(hipMemcpy(dev_, &inner_, sizeof(cuda::DeviceMap), hipMemcpyHostToDevice));
}

void DeviceMapMemWrapper::update_params(const cuda::DeviceMap &existing_map)
{
  // copy inner map, params **only**
  CHECK(hipMemcpy(inner_.size_, existing_map.size_, sizeof(rmagine::Pointi), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(inner_.offset_, existing_map.offset_, sizeof(rmagine::Pointi), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(inner_.pos_, existing_map.pos_, sizeof(rmagine::Pointi), hipMemcpyHostToDevice));

  // copy device pointer 
  CHECK(hipMemcpy(dev_, &inner_, sizeof(cuda::DeviceMap), hipMemcpyHostToDevice));
}


void DeviceMapMemWrapper::to_host(cuda::DeviceMap &output, cuda::DeviceMap &inner, const cuda::DeviceMap *dev, size_t n_voxels)
{
  CHECK(hipMemcpy(&inner, dev, sizeof(cuda::DeviceMap), hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(output.size_, inner.size_, sizeof(rmagine::Pointi), hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(output.offset_, inner.offset_, sizeof(rmagine::Pointi), hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(output.data_, inner.data_, n_voxels * sizeof(TSDFEntry), hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(output.pos_, inner.pos_, sizeof(rmagine::Pointi), hipMemcpyDeviceToHost));
}

cuda::DeviceMap* DeviceMapMemWrapper::to_device(const cuda::DeviceMap &input, cuda::DeviceMap &inner, size_t n_voxels)
{
  cuda::DeviceMap* existing_map_dev;
  CHECK(hipMalloc(&inner.size_, sizeof(rmagine::Pointi)));
  CHECK(hipMalloc(&inner.offset_, sizeof(rmagine::Pointi)));
  CHECK(hipMalloc(&inner.data_, n_voxels * sizeof(TSDFEntry)));
  CHECK(hipMalloc(&inner.pos_, sizeof(rmagine::Pointi)));
  CHECK(hipMemcpy(inner.size_, input.size_, sizeof(rmagine::Pointi), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(inner.offset_, input.offset_, sizeof(rmagine::Pointi), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(inner.data_, input.data_, n_voxels * sizeof(TSDFEntry), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(inner.pos_, input.pos_, sizeof(rmagine::Pointi), hipMemcpyHostToDevice));

  CHECK(hipMalloc((void **) &existing_map_dev, sizeof(cuda::DeviceMap)));
  CHECK(hipMemcpy(existing_map_dev, &inner, sizeof(cuda::DeviceMap), hipMemcpyHostToDevice));
  return existing_map_dev;
}

void DeviceMapMemWrapper::to_host(const cuda::DeviceMap &existing_map)
{
  CHECK(hipMemcpy(&inner_, dev_, sizeof(cuda::DeviceMap), hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(existing_map.size_, inner_.size_, sizeof(rmagine::Pointi), hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(existing_map.offset_, inner_.offset_, sizeof(rmagine::Pointi), hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(existing_map.data_, inner_.data_, n_voxels_ * sizeof(TSDFEntry), hipMemcpyDeviceToHost));
  CHECK(hipMemcpy(existing_map.pos_, inner_.pos_, sizeof(rmagine::Pointi), hipMemcpyDeviceToHost));
}

cuda::DeviceMap* DeviceMapMemWrapper::dev() const
{
  return dev_;
}

} // end namespace cuda
