#include "warpsense/cuda/cleanup.h"

void cuda::pause()
{
  hipDeviceSynchronize();
}

void cuda::cleanup()
{
  hipDeviceReset();
}
